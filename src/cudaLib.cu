#include "hip/hip_runtime.h"

#include "cudaLib.cuh"
#include <thrust/device_vector.h>

void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__global__ 
void saxpy_gpu (float* x, float* y, float scale, int size) {
	//	Insert GPU SAXPY kernel code here
}

int runGpuSaxpy(int vectorSize) {

	std::cout << "Hello GPU Saxpy!\n";

	//	Insert code here
	std::cout << "Lazy, you are!\n";
	std::cout << "Write code, you must\n";

	return 0;
}

__global__
void generatePoints (uint64_t * pSums, uint64_t pSumSize, uint64_t sampleSize) {
	//	Insert code here
}

__global__ 
void reduceCounts (uint64_t * pSums, uint64_t * totals, uint64_t pSumSize, uint64_t reduceSize) {
	//	Insert code here
}

int runGpuMCPi (uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {

	//  Check CUDA device presence
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		return -1;
	}

	auto tStart = std::chrono::high_resolution_clock::now();
		
	float approxPi = estimatePi(generateThreadCount, sampleSize, 
		reduceThreadCount, reduceSize);
	
	std::cout << "Estimated Pi = " << approxPi << "\n";

	auto tEnd= std::chrono::high_resolution_clock::now();

	std::chrono::duration<double> time_span = (tEnd- tStart);
	std::cout << "It took " << time_span.count() << " seconds.";

	return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize, 
	uint64_t reduceThreadCount, uint64_t reduceSize) {
	
	double approxPi = 3.14159f;

	std::cout << "Sneaky, you are ...\n";
	std::cout << "Compute pi, you must!\n";
	return approxPi;
}


#define BLOCK_SIZE 16
#define MAX_WINDOW_SIZE 64

void query_device(){

	int maxThreadPerBlock, maxBlockDim, maxGridDim;
	int maxSharedMemoryPerBlock, maxWarpSize;
	
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		std::cout << "CUDA device missing!\n";
		// return -1;
	}
	else{

		hipError_t err_ThreadPerBlock = hipDeviceGetAttribute(&maxThreadPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
 		// printf("err = %d, %s\n", err_ThreadPerBlock, hipGetErrorString(err_ThreadPerBlock));
  		printf("maxThreadPerBlock = %d\n", maxThreadPerBlock);
		
		hipError_t err_BlockDim = hipDeviceGetAttribute(&maxBlockDim, hipDeviceAttributeMaxBlockDimX, 0);
 		// printf("err = %d, %s\n", err_BlockDim, hipGetErrorString(err_BlockDim));
  		printf("maxBlockDim = %d\n", maxBlockDim);

		hipError_t err_sm = hipDeviceGetAttribute(&maxSharedMemoryPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
 		// printf("err = %d, %s\n", err_BlockDim, hipGetErrorString(err_BlockDim));
  		printf("maxSharedMemoryPerBlock = %d\n", maxSharedMemoryPerBlock);

		hipError_t err_GridDim = hipDeviceGetAttribute(&maxGridDim, hipDeviceAttributeMaxGridDimX, 0);
 		// printf("err = %d, %s\n", err_GridDim, hipGetErrorString(err_GridDim));
  		printf("maxGridDim = %d\n", maxGridDim);

		hipError_t err_WarpSize = hipDeviceGetAttribute(&maxWarpSize, hipDeviceAttributeWarpSize, 0);
 		// printf("err = %d, %s\n", err_BlockDim, hipGetErrorString(err_BlockDim));
  		printf("maxWarpSize = %d\n", maxWarpSize);

	}

}

__device__
uint8_t * sortPixels_gpu (uint8_t * array, dim3 arrayDim){

	uint8_t tmp;
	
	for (int i = 0; i < arrayDim.x - 1; i++) {
        for (int j = i+1; j < arrayDim.x; j++) {
            if (array[i] > array[j]) { 
				//Swap Values.
                tmp = array[i];
                array[i] = array[j];
                array[j] = tmp;
			}
		}
	}

	return array;
};


__global__ 
void medianFilter_gpu (uint8_t * inPixels, ImageDim imgDim, uint8_t * outPixels, MedianFilterArgs args) {

	int row_gl = blockDim.y * blockIdx.y + threadIdx.y;
	int col_gl = blockDim.x * blockIdx.x + threadIdx.x;
	int channels_gl = blockDim.z * blockIdx.z + threadIdx.z;

	int count = 0;

	uint32_t inRow, inCol;

	// __shared__ uint8_t window[MAX_WINDOW_SIZE][MAX_WINDOW_SIZE][3];
	uint8_t window[MAX_WINDOW_SIZE];
	uint8_t * sorted_window = window;

	if (col_gl < imgDim.width && row_gl < imgDim.height && channels_gl < 4) {

		for (uint32_t filRow = 0; filRow < args.filterH; ++ filRow) {
			for (uint32_t filCol = 0; filCol < args.filterW; ++ filCol) {
				inRow = row_gl + filRow;
				inCol = col_gl + filCol;
				
				if(inRow >= 0 && inRow < imgDim.height && 
					inCol >= 0 && inCol < imgDim.width){

					window[count] = inPixels[(inRow * imgDim.width + inCol) * imgDim.channels + channels_gl];
					count++;

				}
			}

		}

		dim3 arraydim(count);

		sorted_window = sortPixels_gpu(sorted_window, arraydim);

		outPixels[imgDim.channels*(row_gl* imgDim.width + col_gl) + channels_gl] = sorted_window[(args.filterH * args.filterW) / 2];

	}	
}


int runGpuMedianFilter (std::string imgPath, std::string outPath, MedianFilterArgs args) {

	ImageDim imgDim;

	uint8_t * imgData, * imgData_d;

	int bytesRead = loadBytesImage(imgPath, imgDim, &imgData);
	int imgSize = imgDim.height * imgDim.width * imgDim.channels * imgDim.pixelSize;

	uint8_t * outData_d;
	uint8_t * outData = (uint8_t *) malloc(imgSize);

	hipMalloc(&imgData_d, imgSize * sizeof(uint8_t));
	hipMalloc(&outData_d, imgSize * sizeof(uint8_t));

	hipMemcpy(imgData_d, imgData, imgSize * sizeof(uint8_t) , hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 3);
    dim3 dimGrid(ceil((float)imgDim.width / (float)dimBlock.x), ceil((float)imgDim.height / (float)dimBlock.y));
	medianFilter_gpu<<<dimGrid, dimBlock>>>(imgData_d, imgDim, outData_d, args);

	hipMemcpy(outData, outData_d, imgSize * sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(imgData_d);
	hipFree(outData_d);

	writeBytesImage(outPath, imgDim, outData);

	return 0;
}


__global__ 
void poolLayer_gpu (float * input, TensorShape inShape, float * output, TensorShape outShape, PoolLayerArgs args){

	int row_gl = blockDim.y * blockIdx.y + threadIdx.y;
	int col_gl = blockDim.x * blockIdx.x + threadIdx.x;

	uint32_t row, col;

	if (col_gl < outShape.width && row_gl < outShape.height) {

		//	STUDENT: Assign to first value of pool area
		float poolPick = input[row_gl * args.strideH * inShape.width + col_gl * args.strideW];

		for (uint32_t poolRow = 0; poolRow < args.poolH; ++ poolRow) {
			for (uint32_t poolCol = 0; poolCol < args.poolW; ++ poolCol) {
				
				//	STUDENT: Calculate row and col of element here

				row = (row_gl * args.strideH) + poolRow;
				col = (col_gl * args.strideW) + poolCol;

				if(row >= 0 && row < inShape.height && 
						col >= 0 && col < inShape.width){

					float value = input[row * inShape.width + col];
					
					switch (args.opType)
					{
					//	STUDENT: Add cases and complete pooling code for all 3 options
					case PoolOp::MaxPool:

						if (value > poolPick)
						{	
							poolPick = value;
						}
						break;

					case PoolOp::MinPool:

						if (value < poolPick)
						{
							poolPick = value;
						}
						break;

					case PoolOp::AvgPool:

						poolPick += value;
						poolPick = poolPick/(args.poolH * args.poolW);
						break;

					default:
						return;	
						break;
					}
				}
			}
		}

		output[row_gl * outShape.width + col_gl] = poolPick;

	}
}


int runGpuPool (TensorShape inShape, PoolLayerArgs poolArgs){
	
	float *input_d, *output_d;
	
	float* inMatrix = (float*) malloc(inShape.height * inShape.width * sizeof(float));

	if (inMatrix == NULL){
		std::cout<< "ERROR ERROR!!!!! RUN FOR THE HILLS!!!!!";
		return 0;
	} 

	TensorShape_t outShape;

	outShape.height = (inShape.height - poolArgs.poolH) / poolArgs.strideH + 1;		
	outShape.width = (inShape.width - poolArgs.poolW) / poolArgs.strideW + 1;
	outShape.channels = inShape.channels;

	std::cout<< "Output Dimensions: " << outShape.height << " * " << outShape.width <<"\n";

	float* outMatrix =  (float*) malloc(outShape.height * outShape.width * sizeof(float));

	if (outMatrix == NULL){
		std::cout<< "ERROR ERROR!!!!! RUN FOR THE HILLS!!!!!";
		return -1;
	} 

	if(hipMalloc(&input_d,  inShape.height * inShape.width * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< inShape.height * inShape.width * sizeof(float);
		std::cout<< "\n ERROR ERROR!!!!! RUN FOR THE HILLS!!!!! INPUT MEMORY ALLOCATION FAILURE \n";
		return -1;
	}
	
	if(hipMalloc(&output_d, outShape.height * outShape.width * sizeof(float))!=hipSuccess){
		std::cout<< "Size Requested: "<< outShape.height * outShape.width * sizeof(float);
		std::cout<< "ERROR ERROR!!!!! RUN FOR THE HILLS!!!!!OUTPUT MEMORY ALLOCATION FAILURE \n";
		return -1;
	}

	for (int i = 0; i < inShape.height; i++){
		for(int j = 0; j < inShape.width; j++){
			inMatrix[i * inShape.width + j] = (rand() / (RAND_MAX + 1.)) * 100;
			// std::cout << inMatrix[i * inShape.width + j] << " ";
		}
		// std::cout << "\n";
	}

	hipMemcpy(input_d, inMatrix, inShape.height * inShape.width * sizeof(float), hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(ceil((float)outShape.width / (float)dimBlock.x), ceil((float)outShape.height / (float)dimBlock.y));

	// STUDENT: call pool function
	poolLayer_gpu<<<dimGrid, dimBlock>>>(input_d, inShape, output_d, outShape, poolArgs);

	hipMemcpy(outMatrix, output_d, outShape.height * outShape.width * sizeof(float), hipMemcpyDeviceToHost);

	// for (int i = 0; i < outShape.height; i++){
	// 	for(int j = 0; j < outShape.width; j++){

	// 		//  std::cout << outMatrix[i*outShape.width + j] << " ";

	// 		// std::cout << outMatrix[i*outShape.width + j] << "@ (" << i << ", " << j << ")" << outShape.width << "\n";
	// 	}
	// 	// std::cout << "\n";
	// }
	
	free(inMatrix);
	free(outMatrix);

	hipFree(output_d);
	hipFree(input_d);

	return 0;
}

